
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  1
#define TPERBLK  33

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

__device__ int lock = 0;

__global__ void kmain(volatile unsigned int *data) 
{
    if(threadIdx.x == 0)
    {
        while(atomicCAS(&lock, 0, 1) != 0) {}
        //__threadfence(); -- Missing
        data[0] = 1;
        __threadfence();
        atomicExch(&lock, 0);
    }
    else if(threadIdx.x == 32)
    {
        while(atomicCAS_block(&lock, 0, 1) != 0) {}
        __threadfence_block();
        data[0] = 2;
        __threadfence_block();
        atomicExch_block(&lock, 0);
    }
}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

