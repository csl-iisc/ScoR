
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  1
#define TPERBLK  33

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}


// @@@code  {Inter warp device-locked vs normal access}   {cuda:wwblocklock}
__device__ int lock = 0;

__global__ void kmain(volatile unsigned int *data)       // @@@{
{
    if(threadIdx.x == 0)
    {
        while(atomicCAS(&lock, 0, 1) != 0) {}
        __threadfence();
        data[0] = 1;
        __threadfence();
        atomicExch(&lock, 0);
    }
    else if(threadIdx.x == 32)
    {
        data[0] = 2;
    }
}                                               // @@@}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

