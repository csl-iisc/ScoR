
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  2
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

// @@@code  {Inter block data write ouside critical section2}   {cuda:bbwrwithoutlock2}
__device__ int lock = 0;

__global__ void kmain(volatile unsigned int *data)       // @@@{
{
    if(blockIdx.x == 0)
    {
        while(atomicCAS(&lock, 0, 1) != 0) {}
        __threadfence();
        data[0] = 1;
        __threadfence();
        atomicExch(&lock, 0);
    }
    else
    {
        data[0] = 2;
    }
}                                               // @@@}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

