
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  2
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}


// @@@code  {Inter block critical sections missing fence}   {cuda:bbnofencelocks}
__device__ int lock = 0;

__global__ void kmain(volatile unsigned int *data)       // @@@{
{
    if(blockIdx.x == 0)
    {
        while(atomicCAS_block(&lock, 0, 1) != 0) {}
        __threadfence_block();
        data[0] = 1;
        __threadfence_block();
        atomicExch_block(&lock, 0);
    }
    else
    {
        while(atomicCAS_block(&lock, 0, 1) != 0) {}
        __threadfence_block();
        data[0] = 2;
        __threadfence_block();
        atomicExch_block(&lock, 0);
    }
}                                               // @@@}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

