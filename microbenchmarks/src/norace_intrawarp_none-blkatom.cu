
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  1
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

__device__ int flag = 0;

__global__ void kmain(volatile unsigned int *data) 
{
    atomicExch_block((unsigned int *)&data[0], 1);
    data[0] = 2;
}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

