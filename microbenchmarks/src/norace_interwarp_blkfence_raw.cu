
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  1
#define TPERBLK  33

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

__device__ int flag = 0;

__global__ void kmain(volatile unsigned int *data) 
{
    if(threadIdx.x == 0)
    {
        data[0] = 1;
        __threadfence_block();
        atomicExch(&flag, 1);
    }
    else if(threadIdx.x == 32)
    {
        while(atomicExch(&flag, 0) == 0) {}
        int a = data[0];
        atomicExch(&flag, a);
    }
}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

