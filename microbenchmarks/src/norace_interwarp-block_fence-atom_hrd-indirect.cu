
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  2
#define TPERBLK  33

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

__device__ int flag = 0;

__global__ void kmain(unsigned int *data) 
{
    if(blockIdx.x == 0 && threadIdx.x == 0)
    {
        atomicExch_block(&data[0], 1);
        __threadfence_block();
        atomicExch(&flag, 1);
    }
    else if(blockIdx.x == 0 && threadIdx.x == 32)
    {
        while(atomicAdd(&flag, 0) != 1);
        atomicExch(&data[0], 2);
        __threadfence();
        atomicExch(&flag, 2);
    }
    else if(blockIdx.x == 1 && threadIdx.x == 0)
    {
        while(atomicAdd(&flag, 0) != 2);
        atomicExch_block(&data[0], 3);
        __threadfence_block();
        atomicExch(&flag, 3);
    }
    else if(blockIdx.x == 1 && threadIdx.x == 32)
    {
        while(atomicAdd(&flag, 0) != 3);
        atomicExch(&data[0], 3);
    }
}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

