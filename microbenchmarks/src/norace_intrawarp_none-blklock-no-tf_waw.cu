
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  1
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

__device__ int lock = 0;

__global__ void kmain(volatile unsigned int *data) 
{
    while(atomicCAS_block(&lock, 0, 1) != 0) {}
    __threadfence_block();
    data[0] = 1;
    atomicExch_block(&lock, 0);
    data[0] = 2;
}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

