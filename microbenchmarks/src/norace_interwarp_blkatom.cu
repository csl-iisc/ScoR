
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  1
#define TPERBLK  33

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}

__device__ int flag = 0;

__global__ void kmain(unsigned int *data) 
{
    if(threadIdx.x == 0)
    {
        atomicExch_block(&data[0], 1);
    }
    else if(threadIdx.x == 32)
    {
        atomicExch_block(&data[0], 2);
    }
}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

