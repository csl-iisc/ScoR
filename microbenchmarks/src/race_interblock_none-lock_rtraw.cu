
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  2
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}


// @@@code  {Inter block data write outside critical section}   {cuda:bbwrwithoutlock}
__device__ int lock = 0;
__device__ int dummy = 0;

__global__ void kmain(volatile unsigned int *data)       // @@@{
{
    if(blockIdx.x == 0)
    {
        while(atomicCAS(&lock, 0, 1) != 0) {}
        __threadfence();
        dummy = data[0];
        __threadfence();
        atomicExch(&lock, 0);
        data[0] = 1;
    }
    else
    {
        while(atomicCAS(&lock, 0, 1) != 0) {}
        __threadfence();
        dummy = data[0];
        __threadfence();
        atomicExch(&lock, 0);
    }
}                                               // @@@}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

