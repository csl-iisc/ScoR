
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  2
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}


// @@@code  {Inter block acquire missing fence}   {cuda:bbnofenceacq}
__device__ int flag = 0;

__global__ void kmain(volatile unsigned int *data)   // @@@{
{
    if(blockIdx.x == 0)
    {
        data[0] = 1;
        __threadfence_block();
        atomicExch(&flag, 1);
    }
    else
    {
        while(atomicExch(&flag, 0) == 0) {}
        int a = data[0];
        atomicExch(&flag, a);
    }
}                                           // @@@}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    errCheck();
    return 0;
}

