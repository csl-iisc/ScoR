#include "hip/hip_runtime.h"
/********************************************************************************************
 * Implementation of Parallel Graph Connectivity Computation
 *
 * Authored by: 
 * Aditya K Kamath, Indian Institute of Science
 *
 *
 * Based on:
 * M. Sutton, T. Ben-Nun and A. Barak
 * Optimizing Parallel Graph Connectivity Computation via Subgraph Sampling
 * 2018 IEEE International Parallel and Distributed Processing Symposium (IPDPS)
 * Original Implementation: https://github.com/michaelsutton/afforest
 *
 * Modified to enable "Work-Stealing" between blocks
 *
 ********************************************************************************************/

#include <iostream>
#include <unordered_set>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
using namespace std;

/**************************************************
 *
 *               KERNEL FUNCTIONS
 *
 **************************************************/

__global__ void initKernel(int *head, int *tail, int *graphComponents, int *bases, int *blockIds)
{
    const int tid = threadIdx.x;
    int bid       = blockIdx.x;
    int *base     = &bases[bid];
    int *blockId  = &blockIds[bid];

    if(tid == 0)
    {
        *base = atomicAdd(&head[bid], NTHREADS);
        atomicExch_block(blockId, bid);
    }

    __syncthreads();

    int my_base = *base;
    int n_t_last = tail[bid];
    while(my_base < n_t_last)
    {
        if(tid + my_base < n_t_last)
        {
            graphComponents[tid + my_base] = tid + my_base;
        }
#ifdef RACEY
#else
        __syncthreads();
#endif

        if(tid == 0) {
            *base = atomicAdd(&head[bid], NTHREADS);
        }

        __syncthreads();
        my_base = *base;

        if(*base < n_t_last)
            continue;

        __syncthreads();
        if (tid == 0)
        {
            int otherBlock = 0;
            for (int block = (bid + 1);
                block < (bid + NBLOCKS); block++) {
                otherBlock = block % NBLOCKS;
                int h = atomicAdd(&head[otherBlock], 0);
                int t = tail[otherBlock];
                if ((h + NTHREADS) < t) {
                    break;
                }
            }
            *base = atomicAdd(&head[otherBlock], NTHREADS);
            atomicExch_block(blockId, otherBlock);
        }
        __syncthreads();
        bid = atomicAdd_block(blockId, 0);
        my_base = *base;
        n_t_last = tail[bid];
    }
}

__global__ void linkKernel(int *head, int *tail, int *edgeSetU, int *edgeSetV, int *graphComponents, int *bases, int *blockIds)
{
    const int tid = threadIdx.x;
    int bid       = blockIdx.x;
    int *base     = &bases[bid];
    int *blockId  = &blockIds[bid];

    if(tid == 0)
    {
        *base = atomicAdd(&head[bid], NTHREADS);
        atomicExch_block(blockId, bid);
    }

    __syncthreads();

    int my_base = *base;
    int n_t_last = tail[bid];
    while(my_base < n_t_last)
    {
        if(tid + my_base < n_t_last)
        {

            int U = edgeSetU[tid + my_base];
            int V = edgeSetV[tid + my_base];
            int p1 = atomicAdd(&graphComponents[U], 0);
            int p2 = atomicAdd(&graphComponents[V], 0);

            while (p1 != p2)
            {
                int maxi = p1 > p2 ? p1 : p2;
                int mini = p1 + (p2 - maxi);

                int prev = atomicCAS(&graphComponents[maxi], maxi, mini);

                if (prev == maxi || prev == mini) 
                    break;

                p1 = atomicAdd(&graphComponents[atomicAdd(&graphComponents[maxi], 0)], 0);
                p2 = atomicAdd(&graphComponents[mini], 0);
            }
        }
#ifdef RACEY
#else
        __syncthreads();
#endif


        if(tid == 0) {
            *base = atomicAdd(&head[bid], NTHREADS);
        }

        __syncthreads();
        my_base = *base;

        if(*base < n_t_last)
            continue;

        __syncthreads();
        if (tid == 0)
        {
            int otherBlock = 0;
            for (int block = (bid + 1);
                block < (bid + NBLOCKS); block++) {
                otherBlock = block % NBLOCKS;
                int h = atomicAdd(&head[otherBlock], 0);
                int t = tail[otherBlock];
                if ((h + NTHREADS) < t) {
                    break;
                }
            }
#ifdef RACEY
            *base = atomicAdd_block(&head[otherBlock], NTHREADS);
#else
            *base = atomicAdd(&head[otherBlock], NTHREADS);
#endif
            atomicExch_block(blockId, otherBlock);
        }
        __syncthreads();
        bid = atomicAdd_block(blockId, 0);
        my_base = *base;
        n_t_last = tail[bid];
    }
}

__global__ void compressKernel(int *head, int *tail, int *graphComponents, int *bases, int *blockIds)
{
    const int tid = threadIdx.x;
    int bid       = blockIdx.x;
    int *base     = &bases[bid];
    int *blockId  = &blockIds[bid];

    if(tid == 0)
    {
        *base = atomicAdd(&head[bid], NTHREADS);
        atomicExch_block(blockId, bid);
    }

    __syncthreads();

    int my_base = *base;
    int n_t_last = tail[bid];
    while(my_base < n_t_last)
    {
        if(tid + my_base < n_t_last)
        {
            int current = atomicAdd(&graphComponents[tid + my_base], 0);
            int parent = atomicAdd(&graphComponents[current], 0);

            while (current != parent)
            {
                int val = atomicCAS(&graphComponents[tid + my_base], current, parent);
                if(val == current)
                {
                    // Successful swap, continue compression
                    current = parent;
                    parent = atomicAdd(&graphComponents[current], 0);
                }
                else
                {
                    // Unsuccessful swap, restart compression
                    current = atomicAdd(&graphComponents[tid + my_base], 0);
                    parent = atomicAdd(&graphComponents[current], 0);
                }
            }
 
        }
#ifdef RACEY
#else
        __syncthreads();
#endif
        if(tid == 0) {
            *base = atomicAdd(&head[bid], NTHREADS);
        }

        __syncthreads();
        my_base = *base;

        if(*base < n_t_last)
            continue;

        __syncthreads();
        if (tid == 0)
        {
            int otherBlock = 0;
            for (int block = (bid + 1);
                block < (bid + NBLOCKS); block++) {
                otherBlock = block % NBLOCKS;
                int h = atomicAdd(&head[otherBlock], 0);
                int t = tail[otherBlock];
                if ((h + NTHREADS) < t) {
                    break;
                }
            }
#ifdef RACEY
            *base = atomicAdd_block(&head[otherBlock], NTHREADS);
#else
            *base = atomicAdd(&head[otherBlock], NTHREADS);
#endif
            atomicExch_block(blockId, otherBlock);
        }
        __syncthreads();
        bid = atomicAdd_block(blockId, 0);
        my_base = *base;
        n_t_last = tail[bid];
    }
}


/**************************************************
 *
 *                HOST FUNCTIONS
 *
 **************************************************/

void errorCheck(hipError_t err, const char location[])
{
    if (err != hipSuccess)
    {
        cout << "Error (" << err <<"): " << hipGetErrorString(err) << "; at " << location << "\n";
        exit(1);
    }
}

/**************************************************
 *              INPUT DESCRIPTION
 *
 * V = Number of vertices
 * E = Number of edges
 *
 * For all i < E
 * where u = edgeListU[i], and v = edgeListV[i]
 * (u, v) is an edge in the graph
 *
 *************************************************/

void input(int &V, int &E, int **edgeListU, int **edgeListV)
{    
    cin >> V >> E;
    (*edgeListU) = new int[E];
    (*edgeListV) = new int[E];
    
    for(int i = 0; i < E; ++i)
    {
        cin >> (*edgeListU)[i] >> (*edgeListV)[i];
    }
}

// Allocate sufficient space on GPU memory
void allocate(int **d_edgeListU,  int **d_edgeListV, int **d_vertexComponent, int **d_head, int **d_tail, int **d_base, int **d_blockId, int V, int E)
{
    errorCheck(hipMalloc((void**)d_edgeListU, sizeof(int) * E), "allocate edgeListU");
    errorCheck(hipMalloc((void**)d_edgeListV, sizeof(int) * E), "allocate edgeListV");
    errorCheck(hipMalloc((void**)d_vertexComponent, sizeof(int) * V), "allocate vertexColor");
    errorCheck(hipMalloc((void**)d_head, sizeof(int) * NBLOCKS), "allocate head");
    errorCheck(hipMalloc((void**)d_tail, sizeof(int) * NBLOCKS), "allocate tail");
    errorCheck(hipMalloc((void**)d_base, sizeof(int) * NBLOCKS), "allocate base");
    errorCheck(hipMalloc((void**)d_blockId, sizeof(int) * NBLOCKS), "allocate blockId");
}

// Divide the edges/vertices amongst the blocks
void divideWork(int *h_head, int *h_tail, int *d_head, int *d_tail, int size, int value)
{
    if(value < size)
    {
        for(int i = 0; i < value; ++i)
        {
            h_head[i] = i;
            h_tail[i] = (i + 1);
        }
    }
    else
    {
        int portion = value / size;
        for(int i = 0; i < size; ++i)
        {
            h_head[i] = i * portion;
            h_tail[i] = (i + 1) * portion;
        }
        h_tail[size - 1] = value;
    }
    errorCheck(hipMemcpy(d_head, h_head, sizeof(int) * NBLOCKS, hipMemcpyHostToDevice), "copy head HtD1");
    errorCheck(hipMemcpy(d_tail, h_tail, sizeof(int) * NBLOCKS, hipMemcpyHostToDevice), "copy tail HtD1");
}

void outputValues(int *d_vertexComponent,  int V)
{
    int *h_vertexComponent = new int[V];
    errorCheck(hipMemcpy(h_vertexComponent, d_vertexComponent, sizeof(int) * V, hipMemcpyDeviceToHost), "copy vertexComponent DtH");
    ofstream out("connectivity-ans.txt");
    unordered_set<int> graphComponents; // Used to count number of components
    for(int i = 0; i < V; ++i)
    {
        out << h_vertexComponent[i] << "\n";
        graphComponents.emplace(h_vertexComponent[i]);
    }

    out << "Total components: " << graphComponents.size() << "\n";
    cout << "Total components: " << graphComponents.size() << "\n";
}

int main()
{
    // Declare and input graph details
    int V, E;
    int *h_edgeListU, *h_edgeListV;
    
    input(V, E, &h_edgeListU, &h_edgeListV);
    
    // Declare other host variables

    int *h_head  = new int[NBLOCKS];
    int *h_tail  = new int[NBLOCKS];
        
    // Declare device variables
    int *d_edgeListU,  *d_edgeListV;
    int *d_vertexComponent;
    int *d_head, *d_tail;
    int *d_base, *d_blockId;

    allocate(&d_edgeListU, &d_edgeListV, &d_vertexComponent, &d_head, &d_tail, &d_base, &d_blockId, V, E);


    // Copy edge list to device
    errorCheck(hipMemcpy(d_edgeListU, h_edgeListU, sizeof(int) * E, hipMemcpyHostToDevice), "copy edgeListU HtD");
    errorCheck(hipMemcpy(d_edgeListV, h_edgeListV, sizeof(int) * E, hipMemcpyHostToDevice), "copy edgeListV HtD");
    
    dim3 dimGrid(NBLOCKS);
    dim3 dimBlock(NTHREADS);

    // Begin connecting graph
    divideWork(h_head, h_tail, d_head, d_tail, NBLOCKS, V);
    initKernel<<<dimGrid, dimBlock>>>(d_head, d_tail, d_vertexComponent, d_base, d_blockId);
    errorCheck(hipDeviceSynchronize(), "initKernel");
    
    divideWork(h_head, h_tail, d_head, d_tail, NBLOCKS, E);
    linkKernel<<<dimGrid, dimBlock>>>(d_head, d_tail, d_edgeListU, d_edgeListV, d_vertexComponent, d_base, d_blockId);
    errorCheck(hipDeviceSynchronize(), "linkKernel");

    divideWork(h_head, h_tail, d_head, d_tail, NBLOCKS, V);
    compressKernel<<<dimGrid, dimBlock>>>(d_head, d_tail, d_vertexComponent, d_base, d_blockId);
    errorCheck(hipDeviceSynchronize(), "compressKernel");    
    
    outputValues(d_vertexComponent, V);

    return 0;
}
