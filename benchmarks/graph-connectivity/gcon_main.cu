/********************************************************************************************
 * Copyright (c) 2020 Indian Institute of Science
 * All rights reserved.
 *
 * Developed by:    Aditya K Kamath
 *                  Computer Systems Lab
 *                  Indian Institute of Science
 *                  https://csl.csa.iisc.ac.in/
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * with the Software without restriction, including without limitation the 
 * rights to use, copy, modify, merge, publish, distribute, sublicense, and/or
 * sell copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 *      > Redistributions of source code must retain the above copyright notice,
 *        this list of conditions and the following disclaimers.
 *      > Redistributions in binary form must reproduce the above copyright
 *        notice, this list of conditions and the following disclaimers in the
 *        documentation and/or other materials provided with the distribution.
 *      > Neither the names of Computer Systems Lab, Indian Institute of Science, 
 *        nor the names of its contributors may be used to endorse or promote products 
 *        derived from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE 
 * CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS WITH
 * THE SOFTWARE.
 *
 ********************************************************************************************/

 /********************************************************************************************
 * Implementation of Parallel Graph Connectivity Computation
 *
 * Based on:
 * M. Sutton, T. Ben-Nun and A. Barak
 * Optimizing Parallel Graph Connectivity Computation via Subgraph Sampling
 * 2018 IEEE International Parallel and Distributed Processing Symposium (IPDPS)
 *
 * Modified to enable "Work-Stealing" between blocks
 *
 ********************************************************************************************/

#include <iostream>
#include <unordered_set>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fstream>
using namespace std;

#include "gcon_kernel.cuh"

/**************************************************
 *
 *                HOST FUNCTIONS
 *
 **************************************************/

void errorCheck(hipError_t err, const char location[])
{
    if (err != hipSuccess) {
        cout << "Error (" << err <<"): " << hipGetErrorString(err) << "; at " << location << "\n";
        exit(1);
    }
}

/**************************************************
 *              INPUT DESCRIPTION
 *
 * V = Number of vertices
 * E = Number of edges
 *
 * For all i < E
 * where u = edgeListU[i], and v = edgeListV[i]
 * (u, v) is an edge in the graph
 *
 *************************************************/

void input(int &V, int &E, int **edgeListU, int **edgeListV)
{    
    cin >> V >> E;
    (*edgeListU) = new int[E];
    (*edgeListV) = new int[E];
    
    for(int i = 0; i < E; ++i) {
        cin >> (*edgeListU)[i] >> (*edgeListV)[i];
    }
}

// Allocate sufficient space on GPU memory
void allocate(int **d_edgeListU,  int **d_edgeListV, int **d_vertexComponent, int **d_head, int **d_tail, int **d_base, int **d_blockId, int V, int E)
{
    errorCheck(hipMalloc((void**)d_edgeListU, sizeof(int) * E), "allocate edgeListU");
    errorCheck(hipMalloc((void**)d_edgeListV, sizeof(int) * E), "allocate edgeListV");
    errorCheck(hipMalloc((void**)d_vertexComponent, sizeof(int) * V), "allocate vertexColor");
    errorCheck(hipMalloc((void**)d_head, sizeof(int) * NBLOCKS), "allocate head");
    errorCheck(hipMalloc((void**)d_tail, sizeof(int) * NBLOCKS), "allocate tail");
    errorCheck(hipMalloc((void**)d_base, sizeof(int) * NBLOCKS), "allocate base");
    errorCheck(hipMalloc((void**)d_blockId, sizeof(int) * NBLOCKS), "allocate blockId");
}

// Divide the edges/vertices amongst the blocks
void divideWork(int *h_head, int *h_tail, int *d_head, int *d_tail, int size, int value)
{
    if(value < size) {
        for(int i = 0; i < value; ++i) {
            h_head[i] = i;
            h_tail[i] = (i + 1);
        }
    }
    else {
        int portion = value / size;
        for(int i = 0; i < size; ++i) {
            h_head[i] = i * portion;
            h_tail[i] = (i + 1) * portion;
        }
        h_tail[size - 1] = value;
    }
    errorCheck(hipMemcpy(d_head, h_head, sizeof(int) * NBLOCKS, hipMemcpyHostToDevice), "copy head HtD1");
    errorCheck(hipMemcpy(d_tail, h_tail, sizeof(int) * NBLOCKS, hipMemcpyHostToDevice), "copy tail HtD1");
}

void outputValues(int *d_vertexComponent,  int V)
{
    int *h_vertexComponent = new int[V];
    errorCheck(hipMemcpy(h_vertexComponent, d_vertexComponent, sizeof(int) * V, hipMemcpyDeviceToHost), "copy vertexComponent DtH");
    ofstream out("connectivity-ans.txt");
    unordered_set<int> graphComponents; // Used to count number of components
    for(int i = 0; i < V; ++i) {
        out << h_vertexComponent[i] << "\n";
        graphComponents.emplace(h_vertexComponent[i]);
    }

    out << "Total components: " << graphComponents.size() << "\n";
    cout << "Total components: " << graphComponents.size() << "\n";
}

int main()
{
    // Declare and input graph details
    int V, E;
    int *h_edgeListU, *h_edgeListV;
    
    input(V, E, &h_edgeListU, &h_edgeListV);
    
    // Declare other host variables

    int *h_head  = new int[NBLOCKS];
    int *h_tail  = new int[NBLOCKS];
        
    // Declare device variables
    int *d_edgeListU,  *d_edgeListV;
    int *d_vertexComponent;
    int *d_head, *d_tail;
    int *d_base, *d_blockId;

    allocate(&d_edgeListU, &d_edgeListV, &d_vertexComponent, &d_head, &d_tail, &d_base, &d_blockId, V, E);

    // Copy edge list to device
    errorCheck(hipMemcpy(d_edgeListU, h_edgeListU, sizeof(int) * E, hipMemcpyHostToDevice), "copy edgeListU HtD");
    errorCheck(hipMemcpy(d_edgeListV, h_edgeListV, sizeof(int) * E, hipMemcpyHostToDevice), "copy edgeListV HtD");
    
    dim3 dimGrid(NBLOCKS);
    dim3 dimBlock(NTHREADS);

    // Begin connecting graph
    divideWork(h_head, h_tail, d_head, d_tail, NBLOCKS, V);
    initKernel<<<dimGrid, dimBlock>>>(d_head, d_tail, d_vertexComponent, d_base, d_blockId);
    errorCheck(hipDeviceSynchronize(), "initKernel");
    
    divideWork(h_head, h_tail, d_head, d_tail, NBLOCKS, E);
    linkKernel<<<dimGrid, dimBlock>>>(d_head, d_tail, d_edgeListU, d_edgeListV, d_vertexComponent, d_base, d_blockId);
    errorCheck(hipDeviceSynchronize(), "linkKernel");

    divideWork(h_head, h_tail, d_head, d_tail, NBLOCKS, V);
    compressKernel<<<dimGrid, dimBlock>>>(d_head, d_tail, d_vertexComponent, d_base, d_blockId);
    errorCheck(hipDeviceSynchronize(), "compressKernel");    
    
    outputValues(d_vertexComponent, V);

    return 0;
}
